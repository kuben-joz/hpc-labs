
#include <hip/hip_runtime.h>
#include <iostream>


struct Sphere {
	float red, green, blue;
	float radius;
	float x, y, z;

	__device__ float hit(float bitmapX, float bitmapY, float *colorFalloff) {
		float distX = bitmapX - x;
		float distY = bitmapY - y;

		if (distX * distX + distY * distY < radius * radius) { 
			float distZ = sqrtf(radius * radius - distX * distX - distY * distY);
			*colorFalloff = distZ / sqrtf(radius * radius);
			return distZ + z; 
		}

		return -1;
	}
};

int main() {
    std::cout << sizeof(Sphere) << std::endl;
    return 0;
}
