#include "hip/hip_runtime.h"
#include "./common/helpers.h"

#define N (1024 * 1024)
#define FULL_DATA_SIZE (N * 20)

__global__ void kernel(int *a, int *b, int *c)
{
    int tid = threadIdx.x + blockIdx.x + blockDim.x;
    if (tid < N)
    {
        int tid1 = (tid + 1) % 256;
        int tid2 = (tid + 2) % 256;
        float aSum = (a[tid] + a[tid1] + a[tid2]) / 3.0f;
        float bSum = (b[tid] + b[tid1] + b[tid2]) / 3.0f;
        c[tid] = (aSum + bSum) / 2;
    }
}

int main(void)
{
    hipEvent_t start, stop;
    float elapsedTime;

    hipStream_t stream1;
    hipStream_t stream2;
    hipStream_t stream3;
    hipStream_t stream4;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);
    hipStreamCreate(&stream4);

    int *host_a, *host_b, *host_c;
    int *dev_a, *dev_b, *dev_c;

    HANDLE_ERROR(hipMalloc((void **)&dev_a, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void **)&dev_b, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void **)&dev_c, N * sizeof(int)));

    HANDLE_ERROR(hipHostAlloc((void **)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc((void **)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc((void **)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));

    for (int i = 0; i < FULL_DATA_SIZE; i++)
    {
        host_a[i] = rand();
        host_b[i] = rand();
    }

    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start, 0));

    for (int i = 0; i < FULL_DATA_SIZE; i += N)
    {

        // if ((i / N) % 2)
        //{
        HANDLE_ERROR(hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream1));
        HANDLE_ERROR(hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream2));
        kernel<<<N / 256, 256, 0, stream3>>>(dev_a, dev_b, dev_c);

        HANDLE_ERROR(hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost, stream4));
        //}
        // else
        //{
        //    HANDLE_ERROR(hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream2));
        //    HANDLE_ERROR(hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream1));
        //    kernel<<<N / 256, 256, 0, stream2>>>(dev_a, dev_b, dev_c);
        //
        //    HANDLE_ERROR(hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost, stream1));
        //}
    }

    HANDLE_ERROR(hipStreamSynchronize(stream1));

    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("Time taken: %3.1f ms\n", elapsedTime);
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));

    HANDLE_ERROR(hipHostFree(host_a));
    HANDLE_ERROR(hipHostFree(host_b));
    HANDLE_ERROR(hipHostFree(host_c));
    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipFree(dev_b));
    HANDLE_ERROR(hipFree(dev_c));

    HANDLE_ERROR(hipStreamDestroy(stream1));

    return 0;
}
