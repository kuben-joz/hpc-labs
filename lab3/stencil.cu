/**
 * stencil.cu: a simple 1d stencil on GPU and on CPU
 *
 * Implement the basic stencil and make sure it works correctly.
 * Then, play with the code
 * - Experiment with block sizes, various RADIUSes and NUM_ELEMENTS.
 * - Measure the memory transfer time, estimate the effective memory bandwidth.
 * - Estimate FLOPS (floating point operations per second)
 * - Switch from float to double: how the performance changes?
 */


#include <hip/hip_runtime.h>
#include <time.h>
#include <stdio.h>
#include <iostream>
#include <vector>
#include <thread>
#include <fstream>

using namespace std;

#define RADIUS 300
#define NUM_ELEMENTS 1000000

#define THREADS_PER_BLOCK 100

#define NUM_CPU_THREADS 12

static void handleError(hipError_t err, const char *file, int line)
{
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}
#define cudaCheck(err) (handleError(err, __FILE__, __LINE__))

__global__ void stencil_1d(double *in, double *out)
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int i = (bid * THREADS_PER_BLOCK) + tid;
    if (i < 0)
    {
        printf("index too small\n");
    }
    else if (i >= NUM_ELEMENTS)
    {
        printf("index too large\n");
    }
    else
    {
        // printf("all ok %d\n", bid);
        for (int j = max(i - RADIUS, 0); j < min(i + RADIUS + 1, NUM_ELEMENTS); j++)
        {
            out[i] += in[j];
        }
    }
}

void thread_exec(double *in, double *out, int mod, int num_threads)
{
    for (int i = mod; i < NUM_ELEMENTS; i += num_threads)
    {
        for (int j = max(i - RADIUS, 0); j < min(i + RADIUS + 1, NUM_ELEMENTS); j++)
        {
            out[i] += in[j];
        }
    }
}

void cpu_stencil_1d(double *in, double *out)
{
    vector<thread> threads;
    for (int i = 0; i < NUM_CPU_THREADS; i++)
    {
        threads.push_back(move(thread(thread_exec, in, out, i, NUM_CPU_THREADS)));
    }

    for (int i = 0; i < NUM_CPU_THREADS; i++)
    {
        threads[i].join();
    }
}

int main()
{
    ofstream device_transfer_file;
    ofstream mem_transfer_file;
    ofstream cpu_file;
    ofstream gpu_file;

    device_transfer_file.open("device_transfer.txt", ios_base::app);
    mem_transfer_file.open("mem_transfer.txt", ios_base::app);
    cpu_file.open("cpu_time.txt", ios_base::app);
    gpu_file.open("gpu_time.txt", ios_base::app);

    // PUT YOUR CODE HERE - INPUT AND OUTPUT ARRAYS
    static double in[NUM_ELEMENTS];
    static double out[NUM_ELEMENTS];

    for (int i = 0; i < NUM_ELEMENTS; i++)
    {
        in[i] = 42.0;
        out[i] = 0.0;
    }
    double *in_gpu;
    double *out_gpu;

    hipEvent_t start, stop, device_transfer, mem_transfer;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&device_transfer);
    hipEventCreate(&mem_transfer);
    hipEventRecord(start, 0);

    // PUT YOUR CODE HERE - DEVICE MEMORY ALLOCATION
    cudaCheck(hipMalloc((void **)&in_gpu, NUM_ELEMENTS * sizeof(*in)));
    cudaCheck(hipMalloc((void **)&out_gpu, NUM_ELEMENTS * sizeof(*out)));

    cudaCheck(hipMemcpy(in_gpu, in, NUM_ELEMENTS * sizeof(*in), hipMemcpyHostToDevice));
    // not neccesary it seems that cuda arrays are always init to 0
    cudaCheck(hipMemcpy(out_gpu, out, NUM_ELEMENTS * sizeof(*out), hipMemcpyHostToDevice));

    hipEventRecord(device_transfer, 0);
    hipEventSynchronize(device_transfer);
    float deviceElapsedTime;
    hipEventElapsedTime(&deviceElapsedTime, start, device_transfer);
    printf("Mem -> Device transfer time:  %.4f ms\n", deviceElapsedTime);
    hipEventDestroy(device_transfer);

    // PUT YOUR CODE HERE - KERNEL EXECUTION

    stencil_1d<<<NUM_ELEMENTS / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(in_gpu, out_gpu);
    hipDeviceSynchronize();

    cudaCheck(hipPeekAtLastError());

    // PUT YOUR CODE HERE - COPY RESULT FROM DEVICE TO HOST
    hipEventRecord(mem_transfer, 0);
    static double out_gpu_cp[NUM_ELEMENTS];
    cudaCheck(hipMemcpy(out_gpu_cp, out_gpu, NUM_ELEMENTS * sizeof(*out_gpu_cp), hipMemcpyDeviceToHost));

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float memElapsedTime;
    hipEventElapsedTime(&memElapsedTime, mem_transfer, stop);
    printf("Device -> memory transfer time:  %.4f ms\n", memElapsedTime);
    float gpuElapsedTime;
    hipEventElapsedTime(&gpuElapsedTime, start, stop);
    printf("Total GPU execution time:  %.4f ms\n", gpuElapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(mem_transfer);

    // PUT YOUR CODE HERE - FREE DEVICE MEMORY
    cudaCheck(hipFree(in_gpu));
    cudaCheck(hipFree(out_gpu));

    struct timespec cpu_start, cpu_stop;
    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &cpu_start);

    cpu_stencil_1d(in, out);

    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &cpu_stop);
    double cpuElapsedTime = (cpu_stop.tv_sec - cpu_start.tv_sec) * 1e3 + (cpu_stop.tv_nsec - cpu_start.tv_nsec) / 1e6;
    printf("CPU execution time:  %.4f ms\n", cpuElapsedTime);

    // PUT YOUR CODE HERE - VERIFY GPU RESULT EQUALS TO CPU RESULT
    for (int i = 0; i < NUM_ELEMENTS; i++)
    {
        if (out_gpu_cp[i] != out[i])
        {
            cout << "GPU not equal to CPU at element " << i << '\n';
            cout << "GPU val " << out_gpu_cp[i] << '\n';
            cout << "CPU val " << out[i] << '\n';
        }
    }

    device_transfer_file << deviceElapsedTime << '\n';
    mem_transfer_file << memElapsedTime << '\n';
    cpu_file << cpuElapsedTime << '\n';
    gpu_file << gpuElapsedTime << '\n';

    return 0;
}
